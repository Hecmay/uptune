
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

//#define N 1573700//1310720//262144//131072//262144//83886080

//Quantidade de threads por blocos
#define BLOCK_SIZE 32//1//1024//95536
#define nThreadsPerBlock 128//420//128//420 ou 416

#define NFinal (nThreadsPerBlock * 5)

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__device__ int* memoria(int *vetDados, int ElemPorBlocos, int qtdProces){

    __shared__ int vetComp[4096];
    
    int auxGrupoDe32 = (qtdProces * 32);

    int comecoBloco = blockIdx.x * ElemPorBlocos; // onde cada bloco irá comeca
    int qtdElemThread = ElemPorBlocos / blockDim.x;

    int idCompartilhada = threadIdx.x;
    
    int idGlobal = comecoBloco + ((threadIdx.x / 32) * qtdElemThread) + (threadIdx.x - ((threadIdx.x / 32) * 32)) + auxGrupoDe32;

    int i;
    for(i = 0; i < 4096; i += blockDim.x){
        vetComp[idCompartilhada] = vetDados[idGlobal];
        idCompartilhada += blockDim.x;
        idGlobal += (qtdElemThread * 4);
    }
    
    return vetComp;
}

__global__ void subSeqMax(int *vet, int *vetFinal, int ElemPorThread, int n){   
    
    __shared__ int *p; // ponteiro para apontar para o vetor compartilhado
    
    //      M         t_m       S      suf
    int ini_M, fim_M, t_M, ini_S, fim_S, suf; //Variaveis do algoritmo
    t_M = suf = 0;

    int comecoThread = (threadIdx.x * 32);

    int j;
    for(j = 0; j < (n / 4096); j++){ // Quantas vezes terei que processa até chegar no n/blocos sendo que o vet compartilhado é de 4096

        p = memoria(vet,n,j);

        __syncthreads();

            if(threadIdx.x < 128){

                ini_M = fim_M = ini_S = fim_S = comecoThread -1;

                int i;
                for(i = comecoThread -1; i < comecoThread + 32; i++){
                    if(i == fim_M){
                        fim_S++;
                        suf += p[i+1];

                        if(suf < 0){
                            suf = 0;
                            fim_S = -1;
                         }
                    
                        ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

                         if(p[i+1] > 0){
                           fim_M++;
                           t_M += p[i+1];
                           ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
                         }
                    }
                    else{
                         if(suf + p[i+1] > t_M){
                             fim_S++;
                             if(ini_M == -1){
                              fim_S = ini_S = i +1;
                             }

                             suf += p[i+1];
                             ini_M = ini_S;
                             fim_M = fim_S;
                             t_M = suf;
                         }
                         else{
                                if(suf + p[i+1] > 0){
                                    fim_S++;
                                    if(suf == 0){
                                        ini_S = fim_S = i+1;
                                    }

                                    suf += p[i+1];
                                }
                                else{
                                    ini_S = fim_S = i + 2;
                                    suf = 0;
                                }
                        }//else
                }//else
            }// 1* for
        }// If 128  
    }// 2* for

    if(threadIdx.x < 128){
        int idThread = blockIdx.x * blockDim.x + threadIdx.x;

        vetFinal[(idThread * 5)] =  vetFinal[(idThread * 5)+1] = vetFinal[(idThread * 5)+2] = vetFinal[(idThread * 5)+3] =
        vetFinal[(idThread * 5)+4] = -1;

        //Colocando o M
        vetFinal[(idThread * 5)+2] = t_M;

        //Calculando o Prefixo
        int pref_Max, soma_Pref;
        soma_Pref  = 0;
        pref_Max = 0;

        int i;
        if(ini_M > comecoThread -1){
            for(i = 0; i < ini_M; i++){
                soma_Pref += p[i];

                if(soma_Pref > pref_Max){
                    pref_Max = soma_Pref;
                }
            }

            if(pref_Max == 0){
                vetFinal[(idThread * 5)] = 0;
                vetFinal[(idThread * 5)+1] = soma_Pref;
            }
            else{
                vetFinal[(idThread * 5)] = pref_Max; //Prefixo
                vetFinal[(idThread * 5)+1] = soma_Pref - pref_Max; //Numeros negativos
            }
        }

        //Calculo do sufixo
        int suf_Max, soma_Suf;
        soma_Suf = suf_Max = 0;

        if(fim_M < comecoThread + 32){
            for(i = (comecoThread + 32)-1; i > fim_M; i--){
                soma_Suf += p[i];

                if(soma_Suf > suf_Max){
                    suf_Max = soma_Suf;
                }
            }

            if(suf_Max == 0){
                vetFinal[(idThread * 5)+3] = 0; //Sufixo vazio
                vetFinal[(idThread * 5)+4] = suf_Max;//Os Numeros negativos

            }
            else{
                vetFinal[(idThread * 5)+3] = suf_Max;   //Sufixo vazio
                vetFinal[(idThread * 5)+4] = soma_Suf - suf_Max;//Os Numeros negativos
            }
        }
    }//if 128
}

void subSeqMaxFinal(int *vet, int n){

    //      M         t_m       S      suf
    int ini_M, fim_M, t_M, ini_S, fim_S, suf;
    ini_M = fim_M = ini_S = fim_S = -1;

    t_M = suf = 0;

    int i;
    for(i = -1; i < n-1; i++){
        if(i == fim_M){
            fim_S++;
            suf += vet[i+1];

            if(suf < 0){
                suf = 0;
                fim_S = -1;
            }

            ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

              if(vet[i+1] > 0){
                   fim_M++;
                   t_M += vet[i+1];
                   ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
               }
        }
        else{
             if(suf + vet[i+1] > t_M){
                 fim_S++;
                 if(ini_M == -1){
                  fim_S = ini_S = i +1;
                 }

                 suf += vet[i+1];
                 ini_M = ini_S;
                 fim_M = fim_S;
                 t_M = suf;

             }
             else{
                    if(suf + vet[i+1] > 0){
                        fim_S++;
                        if(suf == 0){
                            ini_S = fim_S = i+1;
                        }


                    }
                    else{
                        ini_S = fim_S = i + 2;
                        suf = 0;
                    }
             }
        }
    }
    printf("Assertion started\n");
    assert (t_M == 964);
    printf("Assertion Finished");

    printf(" \n\n A sub Sequencia deu %d  \n\n", t_M);
}

int main(int argc, char** argv){

    float elapsedTime;    // Tempo
    hipEvent_t start, stop; // Tempo

    //Vetor aux
    int *vet_d; int *vetFinal_d;
    
    if (argc != 3) {
        fprintf(stderr, "Syntax: %s <Vector size Width> <CacheConfL1> \n", argv[0]);
            return EXIT_FAILURE;
    }

    //Vet
    int N = atoi(argv[1]);
    int *vet_h = (int *) malloc(sizeof(int) * N); // Vetor Dados
    int *vetFinal_h = (int *) malloc(sizeof(int) * NFinal);// Vetor Final

    int i;
    for(i = 0; i < N; i++){ // Preenchimento dos dados
         vet_h[i] = -1;
    }

    for(i = 0; i < NFinal; i++){ // Preenchimento dos dados
         vetFinal_h[i] = -1;
    }

    vet_h[131] = 954;
    vet_h[132] = 10;
    
    int devId = 0;
    int CacheConfL1 = atoi(argv[2]);

    checkCuda( hipSetDevice(devId) );
        hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
    printf("Device: %s\n", prop.name);

    //Reservando o espaço na memoria no device
    hipMalloc((void**)&vet_d, N * sizeof(int)); //Vetor de dados
    hipMalloc((void**)&vetFinal_d, NFinal * sizeof(int));// Vetor Final

    //Copiando o vetor de dados para o device
        hipMemcpy(vet_d, vet_h, N * sizeof(int), hipMemcpyHostToDevice);

    int ElemPorBlocos = (N / BLOCK_SIZE);
    int ElemPorThread = (ElemPorBlocos / nThreadsPerBlock);

    if (CacheConfL1 == 1){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(subSeqMax), hipFuncCachePreferShared);
    }
    else if (CacheConfL1 == 2){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(subSeqMax), hipFuncCachePreferEqual);
    }
    else if (CacheConfL1 == 3){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(subSeqMax), hipFuncCachePreferL1);
    }
    else {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(subSeqMax), hipFuncCachePreferNone);
    }
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipProfilerStart(); 
    subSeqMax<<<BLOCK_SIZE, nThreadsPerBlock>>>(vet_d, vetFinal_d, ElemPorThread,N / BLOCK_SIZE);
    hipProfilerStop();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Primeiro kernel (ms) = %f\n\n", elapsedTime);
    hipMemcpy(vetFinal_h, vetFinal_d, NFinal * sizeof(int), hipMemcpyDeviceToHost); //Resposta Final

    /*for(i = 0; i < 4096; i++){
        if(vetFinal_h[i] != 0 && vetFinal_h[i] != -1 )
            printf("%d ", vetFinal_h[i]);   
    }*/
        
    printf("\n\n");

    hipFree(vetFinal_d);
    hipFree(vet_d);

        subSeqMaxFinal(vetFinal_h, NFinal);

    return 0;
}
